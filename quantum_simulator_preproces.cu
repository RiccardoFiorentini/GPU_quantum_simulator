
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

#define PI  (2*asin(1))
#define GATE_MAX_LEN 63
#define UNITARY_DIM 4

#define GATE_QUBIT "qubit"
#define GATE_CX "cx"
#define GATE_X "x"
#define GATE_SX "sx"
#define GATE_Z "z"
#define GATE_S "s"
#define GATE_SDG "sdg"
#define GATE_T "t"
#define GATE_TDG "tdg"
#define GATE_RZ "rz"
#define GATE_H "h"

#define IS_NOT_CX_OP 127
#define IS_CX_OP 1
#define NUMTHREAD 1024

#define CHECK(call)                                                                       \
{                                                                                     \
    const hipError_t err = call;                                                     \
    if (err != hipSuccess)                                                           \
    {                                                                                 \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE);                                                           \
    }                                                                                 \
}
 
#define CHECK_KERNELCALL()                                                                \
{                                                                                     \
    const hipError_t err = hipGetLastError();                                       \
    if (err != hipSuccess)                                                           \
    {                                                                                 \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE);                                                           \
    }                                                                                 \
}


void putb(long long int, int);
void parse_circuit(char*, int*, int*, float**, float**, char**, char**);

typedef struct{
    float val[4];
}unitary;

typedef struct{
    float val[16];
}unitary4;

//function to get the time of day in seconds
double get_time(){
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

__global__ void init_state_vector(float *vr, float *vi, int num_q){
    int th_id = blockIdx.x*blockDim.x + threadIdx.x;
    if(th_id < (1LLU<<(num_q))){
        vr[th_id] = (th_id==0);
        vi[th_id] = 0.0;
    }
}

//2x2 gates kernel
__global__ void kernel_gate_2(float *vr, float *vi, int num_q, unitary Ur, unitary Ui, int target){
    float tmp0_r, tmp0_i, tmp1_r, tmp1_i;
    int th_id = blockIdx.x*blockDim.x + threadIdx.x;
    long long int pos0, pos1;
    
    if(th_id < (1LLU<<(num_q-1))){
        
        pos0 = ((th_id>>target)<<(target+1))|(th_id&(((1LLU)<<target)-1));
        pos1 = pos0|((1LLU)<<target);

        tmp0_r = vr[pos0]*Ur.val[0] - vi[pos0]*Ui.val[0] + vr[pos1]*Ur.val[1] - vi[pos1]*Ui.val[1];
        tmp0_i = vr[pos0]*Ui.val[0] + vi[pos0]*Ur.val[0] + vr[pos1]*Ui.val[1] + vi[pos1]*Ur.val[1];

        tmp1_r = vr[pos0]*Ur.val[2] - vi[pos0]*Ui.val[2] + vr[pos1]*Ur.val[3] - vi[pos1]*Ui.val[3];
        tmp1_i = vr[pos0]*Ui.val[2] + vi[pos0]*Ur.val[2] + vr[pos1]*Ui.val[3] + vi[pos1]*Ur.val[3];

        vr[pos0] = tmp0_r;
        vr[pos1] = tmp1_r;
        vi[pos0] = tmp0_i;
        vi[pos1] = tmp1_i;
    }
}

//cnot gate kernel
__global__ void kernel_cnot(float *vr, float *vi, int num_q, int control, int target){
    float tmp0_r, tmp0_i, tmp1_r, tmp1_i;
    int th_id = blockIdx.x*blockDim.x + threadIdx.x;
    long long int pos0, pos1;

    int min_idx, max_idx;
    if(th_id < (1LLU<<(num_q-2))){
        min_idx = control < target ? control : target;
        max_idx = control > target ? control : target;

        pos0 = ((th_id>>(max_idx-1))<<(max_idx+1)) | (((th_id&(((1LLU)<<(max_idx-1))-1))>>min_idx)<<(min_idx+1)) | (th_id&(((1LLU)<<min_idx)-1)) | (((1LLU)<<control));
        pos1 = pos0|((1LLU)<<target);

        tmp0_r = vr[pos1];
        tmp0_i = vi[pos1];

        tmp1_r = vr[pos0];
        tmp1_i = vi[pos0];

        vr[pos0] = tmp0_r;
        vr[pos1] = tmp1_r;
        vi[pos0] = tmp0_i;
        vi[pos1] = tmp1_i;

    }
}

void mm2x2(unitary *m1_r, unitary *m2_r, unitary *m1_i, unitary *m2_i){
    unitary tmp_r;
    unitary tmp_i;
    
    tmp_r.val[0] = m1_r->val[0] * m2_r->val[0] - m1_i->val[0] * m2_i->val[0] + m1_r->val[1] * m2_r->val[2] - m1_i->val[1] * m2_i->val[2];
    tmp_i.val[0] = m1_r->val[0] * m2_i->val[0] + m1_i->val[0] * m2_r->val[0] + m1_r->val[1] * m2_i->val[2] + m1_i->val[1] * m2_r->val[2];

    tmp_r.val[1] = m1_r->val[0] * m2_r->val[1] - m1_i->val[0] * m2_i->val[1] + m1_r->val[1] * m2_r->val[3] - m1_i->val[1] * m2_i->val[3];
    tmp_i.val[1] = m1_r->val[0] * m2_i->val[1] + m1_i->val[0] * m2_r->val[1] + m1_r->val[1] * m2_i->val[3] + m1_i->val[1] * m2_r->val[3];

    tmp_r.val[2] = m1_r->val[2] * m2_r->val[0] - m1_i->val[2] * m2_i->val[0] + m1_r->val[3] * m2_r->val[2] - m1_i->val[3] * m2_i->val[2];
    tmp_i.val[2] = m1_r->val[2] * m2_i->val[0] + m1_i->val[2] * m2_r->val[0] + m1_r->val[3] * m2_i->val[2] + m1_i->val[3] * m2_r->val[2];

    tmp_r.val[3] = m1_r->val[2] * m2_r->val[1] - m1_i->val[2] * m2_i->val[1] + m1_r->val[3] * m2_r->val[3] - m1_i->val[3] * m2_i->val[3];
    tmp_i.val[3] = m1_r->val[2] * m2_i->val[1] + m1_i->val[2] * m2_r->val[1] + m1_r->val[3] * m2_i->val[3] + m1_i->val[3] * m2_r->val[3];

    memcpy(m2_r, &tmp_r, sizeof(unitary));
    memcpy(m2_i, &tmp_i, sizeof(unitary));
}

void initM2(unitary *m_r, unitary *m_i){
    m_r->val[0]=1;
    m_r->val[1]=0;
    m_r->val[2]=0;
    m_r->val[3]=1;

    m_i->val[0]=0;
    m_i->val[1]=0;
    m_i->val[2]=0;
    m_i->val[3]=0;
}

bool isIdentity(unitary *m_r, unitary *m_i){
    return fabs(m_r->val[0]-1)<1e-3 && fabs(m_r->val[1])<1e-3 && fabs(m_r->val[2])<1e-3 && fabs(m_r->val[3]-1)<1e-3 && 
            fabs(m_i->val[0])<1e-3 && fabs(m_i->val[1])<1e-3 && fabs(m_i->val[2])<1e-3 && fabs(m_i->val[3])<1e-3;
}

int main(int argc, char *argv[]){

    int num_q, num_g, num_m;
    double *cumul;
    long long meas;
    float *gate_r, *gate_i, *d_state_vec_r, *d_state_vec_i;
    char *target, *cnot_arg;
    unitary Ur, Ui;
    float tmpFloat = 1;
    double t_start, t_end, t_exe;
    unitary *acc_r;
    unitary *acc_i;
    float *sv_r, *sv_i;
    

    if(argc < 2){
        printf("QUANTUM CIRCUIT SIMULATOR\n");
        printf("Usage: %s <circuit_file_name>\n",argv[0]);
        exit(1);
    }

    t_start = get_time();

    parse_circuit(argv[1], &num_q, &num_g, &gate_r, &gate_i, &target, &cnot_arg);
    
    acc_r = (unitary*) malloc(sizeof(unitary)*num_q);
    acc_i = (unitary*) malloc(sizeof(unitary)*num_q);

    for(int i=0; i<num_q; i++){
        initM2(&acc_r[i], &acc_i[i]);
    }

    sv_r = (float*) malloc(sizeof(float)*((1LLU)<<num_q));
    sv_i = (float*) malloc(sizeof(float)*((1LLU)<<num_q));

    CHECK(hipMalloc(&d_state_vec_r, ((1LLU)<<num_q)*sizeof(float)));
    CHECK(hipMalloc(&d_state_vec_i, ((1LLU)<<num_q)*sizeof(float)));
    hipDeviceSynchronize();

    int numBlocks;
    numBlocks = ceil((1LLU<<(num_q))/(double)NUMTHREAD);
    init_state_vector<<<numBlocks, NUMTHREAD>>>(
        d_state_vec_r,
        d_state_vec_i,
        num_q
    );

    hipDeviceSynchronize();
    CHECK_KERNELCALL();

    for(int i=0; i<num_g; i++){
        if(cnot_arg[i]==IS_NOT_CX_OP){
            memcpy(&Ur.val, &(gate_r[i*4]), sizeof(float)*4);
            memcpy(&Ui.val, &(gate_i[i*4]), sizeof(float)*4);
            mm2x2(&Ur, &acc_r[target[i]], &Ui, &acc_i[target[i]]);
        }else{
            numBlocks = ceil((1LLU<<(num_q-1))/(double)NUMTHREAD);
            if(!isIdentity(&acc_r[target[i]], &acc_i[target[i]])){
                kernel_gate_2<<<numBlocks, NUMTHREAD>>>(
                    d_state_vec_r,
                    d_state_vec_i,
                    num_q,
                    acc_r[target[i]],
                    acc_i[target[i]],
                    (int)target[i]
                );
                initM2(&acc_r[target[i]], &acc_i[target[i]]);
            }
            
            if(!isIdentity(&acc_r[cnot_arg[i]], &acc_i[cnot_arg[i]])){
                kernel_gate_2<<<numBlocks, NUMTHREAD>>>(
                    d_state_vec_r,
                    d_state_vec_i,
                    num_q,
                    acc_r[cnot_arg[i]],
                    acc_i[cnot_arg[i]],
                    (int)cnot_arg[i]
                );
                initM2(&acc_r[cnot_arg[i]], &acc_i[cnot_arg[i]]);
            }

            numBlocks = ceil((1LLU<<(num_q-2))/(double)NUMTHREAD);
            kernel_cnot<<<numBlocks, NUMTHREAD>>>(
                d_state_vec_r,
                d_state_vec_i,
                num_q,
                (int)target[i],
                (int)cnot_arg[i]
            );
        }
    }

    numBlocks = ceil((1LLU<<(num_q-1))/(double)NUMTHREAD);
    for(int i=0; i<num_q; i++){
        if(!isIdentity(&acc_r[i], &acc_i[i])){
            kernel_gate_2<<<numBlocks, NUMTHREAD>>>(
                d_state_vec_r,
                d_state_vec_i,
                num_q,
                acc_r[i],
                acc_i[i],
                i
            );
        }
    }

    hipDeviceSynchronize();

    free(acc_i);
    free(acc_r);

    CHECK(hipMemcpy(sv_r, d_state_vec_r, ((1LLU)<<num_q)*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(sv_i, d_state_vec_i, ((1LLU)<<num_q)*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_state_vec_i));
    CHECK(hipFree(d_state_vec_r));

    t_end = get_time();
    t_exe = t_end - t_start;
    
    free(gate_r);
    free(gate_i);
    free(target);
    free(cnot_arg);

    free(sv_r);
    free(sv_i);
   
    //printf("Execution time: %lf\n", t_exe);
    printf("%lf\n", t_exe);

    return 0;
}

void parse_circuit(char *filename, int *num_q, int *num_g, float **gate_r, float **gate_i, char **target, char **cnot_arg){
    FILE *f;
    char c;
    int qubit_num = 0;
    int curr_qubit,curr_qubit2;
    char gate_name[GATE_MAX_LEN+1];
    int str_l;
    float arg;

    f = fopen(filename,"r");
    if(!f){
        printf("ERROR: cannot open circuit file\n");
        exit(1);
    }
    
    fscanf(f,"%d",num_q);
    fscanf(f,"%d",num_g);

    (*gate_r) = (float*) malloc(sizeof(float)*(*num_g)*4);
    (*gate_i) = (float*) malloc(sizeof(float)*(*num_g)*4);
    (*target) = (char*) malloc(sizeof(char)*(*num_g));
    (*cnot_arg) = (char*) malloc(sizeof(char)*(*num_g));

    if(!(*gate_r) || !(*gate_i) || !(*cnot_arg)){
        printf("ERROR: cannot allocate circuit\n");
        free(*gate_r);
        free(*gate_i);
        free(*target);
        free(*cnot_arg);
        exit(1);
    }

    int i;
    fscanf(f,"%c",&c);
    for(i=0; i<(*num_g) && !feof(f); i++){
        while((isblank(c) || c=='\n' || c=='\r' || c==',' || c==';' || !isgraph(c)) && !feof(f)){
            fscanf(f,"%c",&c);
        }

        gate_name[0] = c;
        gate_name[1] = '\0';
        str_l = 1;
        fscanf(f,"%c",&c);
        while(isgraph(c) && c!='[' && str_l<GATE_MAX_LEN){
            gate_name[str_l] = c;
            gate_name[str_l+1] = '\0';
            str_l++;
            fscanf(f,"%c",&c);
        }

        (*cnot_arg)[i] = IS_NOT_CX_OP;
        if(!strcmp(gate_name,GATE_CX)){
            (*cnot_arg)[i] = IS_CX_OP;
        }else if(!strcmp(gate_name,GATE_X)){
            (*gate_r)[i*4]   =  0.0;
            (*gate_r)[i*4+1] =  1.0;
            (*gate_r)[i*4+2] =  1.0;
            (*gate_r)[i*4+3] =  0.0;

            (*gate_i)[i*4]   =  0.0;
            (*gate_i)[i*4+1] =  0.0;
            (*gate_i)[i*4+2] =  0.0;
            (*gate_i)[i*4+3] =  0.0;
        }else if(!strcmp(gate_name,GATE_SX)){
            (*gate_r)[i*4]   =  0.5;
            (*gate_r)[i*4+1] =  0.5;
            (*gate_r)[i*4+2] =  0.5;
            (*gate_r)[i*4+3] =  0.5;

            (*gate_i)[i*4]   =  0.5;
            (*gate_i)[i*4+1] = -0.5;
            (*gate_i)[i*4+2] = -0.5;
            (*gate_i)[i*4+3] =  0.5;
        }else if(!strcmp(gate_name,GATE_Z)){
            (*gate_r)[i*4]   =  1.0;
            (*gate_r)[i*4+1] =  0.0;
            (*gate_r)[i*4+2] =  0.0;
            (*gate_r)[i*4+3] = -1.0;

            (*gate_i)[i*4]   =  0.0;
            (*gate_i)[i*4+1] =  0.0;
            (*gate_i)[i*4+2] =  0.0;
            (*gate_i)[i*4+3] =  0.0;
        }else if(!strcmp(gate_name,GATE_S)){
            (*gate_r)[i*4]   =  1.0;
            (*gate_r)[i*4+1] =  0.0;
            (*gate_r)[i*4+2] =  0.0;
            (*gate_r)[i*4+3] =  0.0;

            (*gate_i)[i*4]   =  0.0;
            (*gate_i)[i*4+1] =  0.0;
            (*gate_i)[i*4+2] =  0.0;
            (*gate_i)[i*4+3] =  1.0;
        }else if(!strcmp(gate_name,GATE_SDG)){
            (*gate_r)[i*4]   =  1.0;
            (*gate_r)[i*4+1] =  0.0;
            (*gate_r)[i*4+2] =  0.0;
            (*gate_r)[i*4+3] =  0.0;

            (*gate_i)[i*4]   =  0.0;
            (*gate_i)[i*4+1] =  0.0;
            (*gate_i)[i*4+2] =  0.0;
            (*gate_i)[i*4+3] = -1.0;
        }else if(!strcmp(gate_name,GATE_T)){
            (*gate_r)[i*4]   =  1.0;
            (*gate_r)[i*4+1] =  0.0;
            (*gate_r)[i*4+2] =  0.0;
            (*gate_r)[i*4+3] =  cos(PI/4);

            (*gate_i)[i*4]   =  0.0;
            (*gate_i)[i*4+1] =  0.0;
            (*gate_i)[i*4+2] =  0.0;
            (*gate_i)[i*4+3] =  sin(PI/4);
        }else if(!strcmp(gate_name,GATE_TDG)){
            (*gate_r)[i*4]   =  1.0;
            (*gate_r)[i*4+1] =  0.0;
            (*gate_r)[i*4+2] =  0.0;
            (*gate_r)[i*4+3] =  cos(PI/4);

            (*gate_i)[i*4]   =  0.0;
            (*gate_i)[i*4+1] =  0.0;
            (*gate_i)[i*4+2] =  0.0;
            (*gate_i)[i*4+3] =  -sin(PI/4);
        }else if(gate_name[0] == GATE_RZ[0] && gate_name[1] == GATE_RZ[1]){
            sscanf(gate_name+3,"%f",&arg);
            (*gate_r)[i*4]   =  1.0;
            (*gate_r)[i*4+1] =  0.0;
            (*gate_r)[i*4+2] =  0.0;
            (*gate_r)[i*4+3] =  cos(arg);

            (*gate_i)[i*4]   =  0.0;
            (*gate_i)[i*4+1] =  0.0;
            (*gate_i)[i*4+2] =  0.0;
            (*gate_i)[i*4+3] =  sin(arg);
        }else if(!strcmp(gate_name,GATE_H)){
            (*gate_r)[i*4]   =  1.0/sqrt(2);
            (*gate_r)[i*4+1] =  1.0/sqrt(2);
            (*gate_r)[i*4+2] =  1.0/sqrt(2);
            (*gate_r)[i*4+3] = -1.0/sqrt(2);

            (*gate_i)[i*4]   =  0.0;
            (*gate_i)[i*4+1] =  0.0;
            (*gate_i)[i*4+2] =  0.0;
            (*gate_i)[i*4+3] =  0.0;
        }else{
            printf("Unknown token: %s\n",gate_name);
            printf("Input format: \n\n");
            printf("<num_qubit> <num_gates>\n");
            printf("<quantum_circuit> \\\\single quantum register\n\n");
            printf("Supported operations: cx, x, sx, z, s, sdg, t, tdg, rz, h\n");
            fclose(f);
            free(*gate_r);
            free(*gate_i);
            free(*target);
            free(*cnot_arg);

            exit(1);
        }
        
        while((c!='$' && c!='[') && !feof(f))
            fscanf(f,"%c",&c);
        fscanf(f,"%d",(*target)+i);

        if((*cnot_arg)[i] == IS_CX_OP){
            fscanf(f,"%c",&c);
            while((c!='$' && c!='[') && !feof(f))
                fscanf(f,"%c",&c);
            fscanf(f,"%d",(*cnot_arg)+i);
        }

        fscanf(f,"%c",&c);
        while((isblank(c) || c==10 || c==',' || c==';' || c==']' || !isgraph(c)) && !feof(f))
            fscanf(f,"%c",&c);
    }

    fclose(f);
    
    return;
}

void putb(long long int n, int len){
    long long int mask = 1LLU << (len-1);
    int m_len = len;
    while(m_len){
        printf("%d",(n&mask)>>(m_len-1));
        mask >>= 1;
        m_len--;
    }
}
